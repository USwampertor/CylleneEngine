#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include ""
#include "math.h"

#include <FreeImage/FreeImage.h>

#include <stdio.h>

#include <cyTime.h>
#include <cyMath.h>
#include <cyVector3f.h>

using namespace CYLLENE_SDK;

bool
LoadTexture(String texturePath, uint8** output, uint32& width, uint32& height);

/**
* gridDim (dim3)
*   dimensions of grid
* blockDim (dim3)
*   dimensions of block
* blockIdx (uint3)
*   block index within grid
* threadIdx (uint3)
*   thread index within block
*/

__device__ int
GetLinearIndex() {
  int blockId = (gridDim.x * gridDim.y * blockIdx.z) +
                (blockIdx.y * gridDim.x) + blockIdx.x;

  int threadId = ((blockDim.x * blockDim.y * blockDim.z) * blockId) +
                 ((blockDim.x * blockDim.y) * threadIdx.z) +
                 ((threadIdx.x * blockDim.y) + threadIdx.x);
  return threadId;
}

__global__ void
InitializeBuffer(uint32 totalPixels, unsigned int* histogram) {
  unsigned int pixelIndex = (blockDim.x * blockIdx.x) + threadIdx.x;
  
  if (pixelIndex >= totalPixels) return;

  const uint32 IMAGE_CHANNELS = 4;

  const unsigned int channel = blockIdx.y;

  histogram[(pixelIndex * 3) + channel] = 0;
}

__global__ void
Histogram(uint8* bytes, int* histogram, uint32 totalPixels) {
  unsigned int pixelIndex = (blockDim.x * blockIdx.x) + threadIdx.x;

  if (pixelIndex >= totalPixels) return;

  const uint32 IMAGE_CHANNELS = 4;

  const unsigned int channel = blockIdx.y;

  uint8 pixel_channel = bytes[(pixelIndex * IMAGE_CHANNELS) + channel];

  atomicAdd(&histogram[(pixel_channel * 3) + channel], 1);
  //atomicAdd(&histogram[channel], 1);
}

int
main(int argc, char** argv) {
  auto secEnum = DELTA_TYPE::E::SECOND;

  hipError_t cudaStatus;

  uint32 imageWidth;
  uint32 imageHeight;
  uint32 totalPixels;

  uint8* bytes = nullptr;
  int* histogramValues = nullptr;

  std::string currentDirectory(argv[0]);
  currentDirectory = currentDirectory.substr(0, currentDirectory.find_last_of('\\') + 1);
  printf("Current path: %s\n", currentDirectory.c_str());

  {
    uint8* imgBytes = nullptr;
    if (!LoadTexture(currentDirectory + "wench.jpg", &imgBytes, imageWidth, imageHeight)) {
      return -1;
    }

    cudaStatus = hipMallocManaged(&bytes,
                                   imageWidth * imageHeight * 4 * sizeof(uint8));
    if (cudaStatus != hipSuccess) {
      return -1;
    }

    std::memcpy(bytes, imgBytes, imageWidth * imageHeight * 4 * sizeof(uint8));

    delete[] imgBytes;
  }

  totalPixels = imageWidth * imageHeight;

  printf("Image loaded. Size = %dx%d\n", imageWidth, imageHeight);

  cudaStatus = hipMallocManaged(&histogramValues, 256 * sizeof(int) * 3);
  if (cudaStatus != hipSuccess) {
    return -1;
  }

  for (int i = 0; i < 256; ++i) {
    histogramValues[(i * 3) + 0] = 0;
    histogramValues[(i * 3) + 1] = 0;
    histogramValues[(i * 3) + 2] = 0;
  }

  Time timer;
  timer.init();

  dim3 blockSize(1024, 1, 1);
  dim3 dispatchSize(Math::ceil(totalPixels / 1024.0f), 3, 1);

  Histogram<<<dispatchSize, blockSize>>>(bytes, histogramValues, totalPixels);
  cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
  }

  hipDeviceSynchronize();

  timer.update();
  printf("Initialization time: %f\n", timer.deltaTime(secEnum));

  for (int i = 0; i < 256; ++i) {
    int r = histogramValues[(i * 3) + 0];
    int g = histogramValues[(i * 3) + 1];
    int b = histogramValues[(i * 3) + 2];

    printf("Pixel %d: %d, %d, %d\n", i, r, g, b);
  }

  timer.update();
  printf("Time required to compute: %f\n", timer.deltaTime(secEnum));

  hipFree(bytes);
  hipFree(histogramValues);

  return 0;
}


bool
LoadTexture(String texturePath, uint8** output, uint32& width, uint32& height) {
  FREE_IMAGE_FORMAT fif = FreeImage_GetFIFFromFilename(texturePath.c_str());

  uint32 flags = 0;
  FIBITMAP* fileImage = FreeImage_Load(fif, texturePath.c_str(), flags);

  if (fileImage == nullptr) {
    printf("Error while loading image\n");
    return false;
  }

  FIBITMAP* image = FreeImage_ConvertTo32Bits(fileImage);
  FreeImage_Unload(fileImage);

  if (image == nullptr) {
    printf("Error while converting image\n");
    return false;
  }

  BYTE* bits = FreeImage_GetBits(image);

  if (bits == nullptr) {
    printf("Error while getting image bytes\n");
    return false;
  }

  width = FreeImage_GetWidth(image);
  height = FreeImage_GetHeight(image);
  unsigned int bytes = FreeImage_GetBPP(image) / 8;

  (*output) = reinterpret_cast<uint8*>(malloc(width * height * bytes));
  std::memcpy((*output), &bits[0], width * height * bytes);

  FreeImage_Unload(image);
  return true;
}